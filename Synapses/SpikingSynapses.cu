#include "hip/hip_runtime.h"
#include "SpikingSynapses.h"

#include "../Helpers/CUDAErrorCheckHelpers.h"
#include "../Helpers/TerminalHelpers.h"


// SpikingSynapses Constructor
SpikingSynapses::SpikingSynapses() {

	delays = NULL;
	stdp = NULL;

	d_delays = NULL;
	d_spikes_travelling_to_synapse = NULL;
	d_stdp = NULL;
	d_time_of_last_spike_to_reach_synapse = NULL;

	maximum_axonal_delay_in_timesteps = 0;
}

// SpikingSynapses Destructor
SpikingSynapses::~SpikingSynapses() {
	// Just need to free up the memory
	// Full Matrices
	free(delays);
	free(stdp);

	CudaSafeCall(hipFree(d_delays));
	CudaSafeCall(hipFree(d_spikes_travelling_to_synapse));
	CudaSafeCall(hipFree(d_stdp));
	CudaSafeCall(hipFree(d_time_of_last_spike_to_reach_synapse));

}

// Connection Detail implementation
//	INPUT:
//		Pre-neuron population ID
//		Post-neuron population ID
//		An array of the exclusive sum of neuron populations
//		CONNECTIVITY_TYPE (Constants.h)
//		2 number float array for weight range
//		2 number float array for delay range
//		Boolean value to indicate if population is STDP based
//		Parameter = either probability for random synapses or S.D. for Gaussian
void SpikingSynapses::AddGroup(int presynaptic_group_id, 
						int postsynaptic_group_id, 
						Neurons * neurons,
						Neurons * input_neurons,
						float timestep,
						synapse_parameters_struct * synapse_params) {
	
	
	Synapses::AddGroup(presynaptic_group_id, 
							postsynaptic_group_id, 
							neurons,
							input_neurons,
							timestep,
							synapse_params);

	spiking_synapse_parameters_struct * spiking_synapse_group_params = (spiking_synapse_parameters_struct*)synapse_params;

	for (int i = original_number_of_synapses; i < total_number_of_synapses; i++){
		
		// Convert delay range from time to number of timesteps
		int delay_range_in_timesteps[2] = {int(round(spiking_synapse_group_params->delay_range[0]/timestep)), int(round(spiking_synapse_group_params->delay_range[1]/timestep))};

		// Check delay range bounds greater than timestep
		if ((delay_range_in_timesteps[0] < 1) || (delay_range_in_timesteps[1] < 1)) {
			printf("%d\n", delay_range_in_timesteps[0]);
			printf("%d\n", delay_range_in_timesteps[1]);
			print_message_and_exit("Delay range must be at least one timestep.");
		}

		// Setup Delays
		if (delay_range_in_timesteps[0] == delay_range_in_timesteps[1]) {
			delays[i] = delay_range_in_timesteps[0];
		} else {
			float random_delay = delay_range_in_timesteps[0] + (delay_range_in_timesteps[1] - delay_range_in_timesteps[0]) * ((float)rand() / (RAND_MAX));
			delays[i] = round(random_delay);
		}

		// printf("delay_range_in_timesteps[0]: %d\n", delay_range_in_timesteps[0]);
		// printf("delay_range_in_timesteps[1]: %d\n", delay_range_in_timesteps[1]);

		if (delay_range_in_timesteps[0] > maximum_axonal_delay_in_timesteps){
			maximum_axonal_delay_in_timesteps = delay_range_in_timesteps[0];
		} else if (delay_range_in_timesteps[1] > maximum_axonal_delay_in_timesteps){
			maximum_axonal_delay_in_timesteps = delay_range_in_timesteps[1];
		}

		// printf("maximum_axonal_delay_in_timesteps: %d\n", maximum_axonal_delay_in_timesteps);

		//Set STDP on or off for synapse
		stdp[i] = spiking_synapse_group_params->stdp_on;
	}

}

void SpikingSynapses::increment_number_of_synapses(int increment) {

	Synapses::increment_number_of_synapses(increment);

    delays = (int*)realloc(delays, total_number_of_synapses * sizeof(int));
    stdp = (bool*)realloc(stdp, total_number_of_synapses * sizeof(bool));

}


void SpikingSynapses::allocate_device_pointers() {

	Synapses::allocate_device_pointers();

	CudaSafeCall(hipMalloc((void **)&d_delays, sizeof(int)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_stdp, sizeof(bool)*total_number_of_synapses));

	CudaSafeCall(hipMalloc((void **)&d_spikes_travelling_to_synapse, sizeof(int)*total_number_of_synapses));
	CudaSafeCall(hipMalloc((void **)&d_time_of_last_spike_to_reach_synapse, sizeof(float)*total_number_of_synapses));

	
}


void SpikingSynapses::copy_constants_and_initial_efficacies_to_device() {
	
	Synapses::copy_constants_and_initial_efficacies_to_device();

	CudaSafeCall(hipMemcpy(d_delays, delays, sizeof(int)*total_number_of_synapses, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_stdp, stdp, sizeof(bool)*total_number_of_synapses, hipMemcpyHostToDevice));

}


void SpikingSynapses::reset_synapse_activities() {

	Synapses::reset_synapse_activities();
	
	reset_time_related_synapse_activities();

}

void SpikingSynapses::reset_time_related_synapse_activities() {

	CudaSafeCall(hipMemset(d_spikes_travelling_to_synapse, 0, sizeof(int)*total_number_of_synapses));
	// Set last spike times to -1000 so that the times do not affect current simulation.
	float* last_spike_to_reach_synapse;
	last_spike_to_reach_synapse = (float*)malloc(sizeof(float)*total_number_of_synapses);
	for (int i=0; i < total_number_of_synapses; i++){
		last_spike_to_reach_synapse[i] = -1000.0f;
	}
	CudaSafeCall(hipMemcpy(d_time_of_last_spike_to_reach_synapse, last_spike_to_reach_synapse, total_number_of_synapses*sizeof(float), hipMemcpyHostToDevice));

}


void SpikingSynapses::shuffle_synapses() {
	
	Synapses::shuffle_synapses();

	int * temp_delays = (int *)malloc(total_number_of_synapses*sizeof(int));
	bool * temp_stdp = (bool *)malloc(total_number_of_synapses*sizeof(bool));
	for(int i = 0; i < total_number_of_synapses; i++) {

		temp_delays[i] = delays[original_synapse_indices[i]];
		temp_stdp[i] = stdp[original_synapse_indices[i]];

	}

	delays = temp_delays;
	stdp = temp_stdp;

}


void SpikingSynapses::set_threads_per_block_and_blocks_per_grid(int threads) {
	
	Synapses::set_threads_per_block_and_blocks_per_grid(threads);
	
}

void SpikingSynapses::interact_spikes_with_synapses(SpikingNeurons * neurons, SpikingNeurons * input_neurons, float current_time_in_seconds, float timestep) {

	if (neurons->high_fidelity_spike_flag){
		check_bitarray_for_presynaptic_neuron_spikes<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(
								d_presynaptic_neuron_indices,
								d_delays,
								neurons->d_bitarray_of_neuron_spikes,
								input_neurons->d_bitarray_of_neuron_spikes,
								neurons->bitarray_length,
								neurons->bitarray_maximum_axonal_delay_in_timesteps,
								current_time_in_seconds,
								timestep,
								total_number_of_synapses,
								d_time_of_last_spike_to_reach_synapse);
		CudaCheckError();
	}
	else{
		move_spikes_towards_synapses_kernel<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(d_presynaptic_neuron_indices,
																			d_delays,
																			d_spikes_travelling_to_synapse,
																			neurons->d_last_spike_time_of_each_neuron,
																			input_neurons->d_last_spike_time_of_each_neuron,
																			current_time_in_seconds,
																			total_number_of_synapses,
																			d_time_of_last_spike_to_reach_synapse);
		CudaCheckError();
	}
}



void SpikingSynapses::calculate_postsynaptic_current_injection_components(SpikingNeurons * neurons, float current_time_in_seconds, float timestep) {

}

void SpikingSynapses::update_synaptic_conductances(float timestep, float current_time_in_seconds) {

}


void SpikingSynapses::test_calcuate_total_current_injections_synapses_version_kernal(Neurons * neurons) {


calcuate_total_current_injections_synapses_version_kernal4<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(number_of_addition_stages,
																															d_array_of_stage_start_indices,
																															d_array_of_number_of_additions_per_stage,
																															d_array_of_sorted_synapse_indices_for_lhs_of_addition,
																															d_array_of_sorted_synapse_indices_for_rhs_of_addition,
																															d_component_current_injections_for_each_synapse);


	// for (int stage_count = 0; stage_count < number_of_addition_stages; stage_count++) {

	// for (int stage_count = 0; stage_count < 1; stage_count++) {

	// 	calcuate_total_current_injections_synapses_version_kernal3<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(array_of_stage_start_indices[stage_count],
	// 																														array_of_number_of_additions_per_stage[stage_count],
	// 																														d_array_of_sorted_synapse_indices_for_lhs_of_addition,
	// 																														d_array_of_sorted_synapse_indices_for_rhs_of_addition,
	// 																														d_component_current_injections_for_each_synapse);

	// }

	CudaCheckError();


	copy_calculated_current_injections_to_neuron_current_injection_array<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(neurons->total_number_of_neurons,
																			neurons->d_current_injections,
																			neurons->d_postsynaptic_neuron_start_indices_for_sorted_conductance_calculations,
																			d_component_current_injections_for_each_synapse);

	CudaCheckError();


}

__global__ void calcuate_total_current_injections_synapses_version_kernal4(int number_of_addition_stages,
																		int* d_array_of_stage_start_indices,
																		int* d_array_of_number_of_additions_per_stage,
																		int* d_array_of_sorted_synapse_indices_for_lhs_of_addition,
																		int* d_array_of_sorted_synapse_indices_for_rhs_of_addition,
																		float* d_component_current_injections_for_each_synapse) {


	// int start_index_for_stage = d_array_of_stage_start_indices[stage_count];
	int start_index_for_stage = 0;

	for (int stage_count = 0; stage_count < number_of_addition_stages; stage_count++) {
		int number_of_additions_for_stage = d_array_of_number_of_additions_per_stage[stage_count];
		if (stage_count!= 0) {
			start_index_for_stage += number_of_addition_stages;	
		}
		

		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		while (idx < number_of_additions_for_stage) {

			d_component_current_injections_for_each_synapse[d_array_of_sorted_synapse_indices_for_lhs_of_addition[start_index_for_stage+idx]] += d_component_current_injections_for_each_synapse[d_array_of_sorted_synapse_indices_for_rhs_of_addition[start_index_for_stage+idx]];

			idx += blockDim.x * gridDim.x;
		
		}
	// __syncthreads();

	}


}



// __global__ void calcuate_total_current_injections_synapses_version_kernal3(int start_index_for_stage,
// 																		int number_of_additions_for_stage,
// 																		int* d_array_of_sorted_synapse_indices_for_lhs_of_addition,
// 																		int* d_array_of_sorted_synapse_indices_for_rhs_of_addition,
// 																		float* d_component_current_injections_for_each_synapse) {


// 	for (int stage_count = 0; stage_count < 17; stage_count++) {

// 		int idx = threadIdx.x + blockIdx.x * blockDim.x;
// 		while (idx < number_of_additions_for_stage) {

// 			d_component_current_injections_for_each_synapse[d_array_of_sorted_synapse_indices_for_lhs_of_addition[start_index_for_stage+idx]] += d_component_current_injections_for_each_synapse[d_array_of_sorted_synapse_indices_for_rhs_of_addition[start_index_for_stage+idx]];

// 			idx += blockDim.x * gridDim.x;
		
// 		}
// 	__syncthreads();

// 	}


// }


__global__ void copy_calculated_current_injections_to_neuron_current_injection_array(int total_number_of_neurons,
																					float* d_current_injections,
																					int* d_postsynaptic_neuron_start_indices_for_sorted_conductance_calculations,
																					float* d_component_current_injections_for_each_synapse) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < total_number_of_neurons) {

		d_current_injections[idx] = d_component_current_injections_for_each_synapse[d_postsynaptic_neuron_start_indices_for_sorted_conductance_calculations[idx]];

		idx += blockDim.x * gridDim.x;
	}
	__syncthreads();


}




// void SpikingSynapses::test_calcuate_total_current_injections_synapses_version_kernal(Neurons * neurons) {


// 	for (int temp_iteration_index = 0; temp_iteration_index < 5; temp_iteration_index++) {

// 		int iteration_buffer = powf(temp_iteration_index, 2);

// 		// printf("temp_iteration_index: %d\n", temp_iteration_index);
// 		// calcuate_total_current_injections_synapses_version_kernal<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(d_component_current_injections_for_each_synapse,
// 		// 												neurons->d_postsynaptic_neuron_start_indices_for_sorted_conductance_calculations,
// 		// 												total_number_of_synapses,
// 		// 												neurons->d_current_injections,
// 		// 												d_postsynaptic_neuron_indices,
// 		// 												neurons->total_number_of_neurons,
// 		// 												temp_iteration_index);

// 		calcuate_total_current_injections_synapses_version_kernal2<<<number_of_synapse_blocks_per_grid, threads_per_block>>>(d_component_current_injections_for_each_synapse,
// 														neurons->d_postsynaptic_neuron_start_indices_for_sorted_conductance_calculations,
// 														total_number_of_synapses,
// 														neurons->d_current_injections,
// 														d_postsynaptic_neuron_indices,
// 														neurons->total_number_of_neurons,
// 														temp_iteration_index,
// 														d_indices_of_sorted_synapses_in_orginal_arrays,
// 														neurons->d_per_neuron_afferent_synapse_count,
// 														iteration_buffer);

// 		CudaCheckError();

// 	}

// 	// printf("end\n");

// }


// __global__ void calcuate_total_current_injections_synapses_version_kernal2(float* d_component_current_injections_for_each_synapse,
// 														int* d_postsynaptic_neuron_start_indices_for_sorted_conductance_calculations,
// 														int total_number_of_synapses,
// 														float* d_current_injections,
// 														int* d_postsynaptic_neuron_indices,
// 														int total_number_of_neurons,
// 														int temp_iteration_index,
// 														int* d_indices_of_sorted_synapses_in_orginal_arrays,
// 														int* d_per_neuron_afferent_synapse_count,
// 														int iteration_buffer) {

// 	// Get thread IDs
// 	int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
// 	int idx = t_idx;

// 	temp_iteration_index = 0;

// 	total_number_of_synapses = total_number_of_synapses / 2;

// 	while (idx < total_number_of_synapses) {

// 		int sorted_syn1_idx = idx;
// 		int sorted_syn2_idx = sorted_syn1_idx + iteration_buffer;

// 		if (sorted_syn2_idx < total_number_of_synapses) {

// 			int idx_of_syn1_in_original_arrays = d_indices_of_sorted_synapses_in_orginal_arrays[sorted_syn1_idx];
// 			int idx_of_syn2_in_original_arrays = d_indices_of_sorted_synapses_in_orginal_arrays[sorted_syn2_idx];

// 			int syn1_postsyn_neuron_idx = d_postsynaptic_neuron_indices[idx_of_syn1_in_original_arrays];
// 			int syn2_postsyn_neuron_idx = d_postsynaptic_neuron_indices[idx_of_syn2_in_original_arrays];

// 			if (syn1_postsyn_neuron_idx == syn2_postsyn_neuron_idx) {

// 				// int start_idx_for_postsyn_neurons_sorted_syns = d_postsynaptic_neuron_start_indices_for_sorted_conductance_calculations[syn1_postsyn_neuron_idx];
// 				// int total_number_of_afferent_synapses_for_postsyn_neuron = d_per_neuron_afferent_synapse_count[syn1_postsyn_neuron_idx];
// 				// int end_idx_for_postsyn_neurons_sorted_syns = start_idx_for_postsyn_neurons_sorted_syns + total_number_of_afferent_synapses_for_postsyn_neuron;

// 				// if (sorted_syn2_idx < end_idx_for_postsyn_neurons_sorted_syns) {

// 					d_component_current_injections_for_each_synapse[sorted_syn1_idx] += d_component_current_injections_for_each_synapse[sorted_syn2_idx];

// 				// }

// 			}

// 		}

// 		idx += blockDim.x * gridDim.x;

// 	}
// 	__syncthreads();

// }



__global__ void move_spikes_towards_synapses_kernel(int* d_presynaptic_neuron_indices,
								int* d_delays,
								int* d_spikes_travelling_to_synapse,
								float* d_last_spike_time_of_each_neuron,
								float* d_input_neurons_last_spike_time,
								float current_time_in_seconds,
								size_t total_number_of_synapses,
								float* d_time_of_last_spike_to_reach_synapse){

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < total_number_of_synapses) {


		int timesteps_until_spike_reaches_synapse = d_spikes_travelling_to_synapse[idx];
		timesteps_until_spike_reaches_synapse -= 1;

		if (timesteps_until_spike_reaches_synapse == 0) {
			d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
		}

		if (timesteps_until_spike_reaches_synapse < 0) {

			// Get presynaptic neurons last spike time
			int presynaptic_neuron_index = d_presynaptic_neuron_indices[idx];
			bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(presynaptic_neuron_index);
			float presynaptic_neurons_last_spike_time = presynaptic_is_input ? d_input_neurons_last_spike_time[CORRECTED_PRESYNAPTIC_ID(presynaptic_neuron_index, presynaptic_is_input)] : d_last_spike_time_of_each_neuron[presynaptic_neuron_index];

			if (presynaptic_neurons_last_spike_time == current_time_in_seconds){

				timesteps_until_spike_reaches_synapse = d_delays[idx];

			}
		} 

		d_spikes_travelling_to_synapse[idx] = timesteps_until_spike_reaches_synapse;

		idx += blockDim.x * gridDim.x;
	}
	__syncthreads();
}

__global__ void check_bitarray_for_presynaptic_neuron_spikes(int* d_presynaptic_neuron_indices,
								int* d_delays,
								unsigned char* d_bitarray_of_neuron_spikes,
								unsigned char* d_input_neuron_bitarray_of_neuron_spikes,
								int bitarray_length,
								int bitarray_maximum_axonal_delay_in_timesteps,
								float current_time_in_seconds,
								float timestep,
								size_t total_number_of_synapses,
								float* d_time_of_last_spike_to_reach_synapse){
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	while (idx < total_number_of_synapses) {

		int presynaptic_neuron_index = d_presynaptic_neuron_indices[idx];
		bool presynaptic_is_input = PRESYNAPTIC_IS_INPUT(presynaptic_neuron_index);
		int delay = d_delays[idx];

		// Get offset depending upon the current timestep
		int offset_index = ((int)(round(current_time_in_seconds / timestep)) % bitarray_maximum_axonal_delay_in_timesteps) - delay;
		offset_index = (offset_index < 0) ? (offset_index + bitarray_maximum_axonal_delay_in_timesteps) : offset_index;
		int offset_byte = offset_index / 8;
		int offset_bit_pos = offset_index - (8 * offset_byte);

		// Get the correct neuron index
		int neuron_index = CORRECTED_PRESYNAPTIC_ID(presynaptic_neuron_index, presynaptic_is_input);
		
		// Check the spike
		int neuron_id_spike_store_start = neuron_index * bitarray_length;
		int check = 0;
		if (presynaptic_is_input){
			unsigned char byte = d_input_neuron_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
			check = ((byte >> offset_bit_pos) & 1);
			if (check == 1){
				d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
			}
		} else {
			unsigned char byte = d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
			check = ((byte >> offset_bit_pos) & 1);
			if (check == 1){
				d_time_of_last_spike_to_reach_synapse[idx] = current_time_in_seconds;
			}
		}

		idx += blockDim.x * gridDim.x;
	}
	__syncthreads();
}